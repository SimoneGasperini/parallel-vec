#include <iostream>
#include "../../hpp/vector.hpp"
#include "../../include/parallel_func.h"

Vector::Vector(int _size, float *_array, bool _malloc)
{
	size = _size;

	if (_malloc)
	{
		hipMalloc(&array, size * sizeof(float));
		for (int i = 0; i < size; i++)
			array[i] = _array[i];
	}
	else
	{
		array = _array;
	}
}

void Vector::free()
{
	hipFree(array);
}

Vector Vector::operator+(const Vector &that)
{
	float *result;
	hipMalloc(&result, size * sizeof(float));

	int threadsPerBlock = 256;
	int numberOfBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
	parallel_add<<<numberOfBlocks, threadsPerBlock>>>(size, array, that.array, result);
	hipDeviceSynchronize();

	Vector sum(size, result, false);

	return sum;
}

float Vector::operator*(const Vector &that)
{
	float *result;
	hipMalloc(&result, size * sizeof(float));

	int threadsPerBlock = 256;
	int numberOfBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
	parallel_mul<<<numberOfBlocks, threadsPerBlock>>>(size, array, that.array, result);
	hipDeviceSynchronize();

	float dot = 0;
	for (int i = 0; i < size; i++)
		dot += result[i];

	return dot;
}

Vector Vector::operator*(float scalar)
{
	float *result;
	hipMalloc(&result, size * sizeof(float));

	int threadsPerBlock = 256;
	int numberOfBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
	parallel_mul<<<numberOfBlocks, threadsPerBlock>>>(size, array, scalar, result);
	hipDeviceSynchronize();

	Vector prod(size, result, false);

	return prod;
}

std::ostream &operator<<(std::ostream &os, Vector &v)
{
	os << "[";
	for (int i = 0; i < v.size; i++)
	{
		os << v.array[i];
		if (i != v.size - 1)
			os << "  ";
	}
	os << "]";

	return os;
}