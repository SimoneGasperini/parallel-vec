#include "hip/hip_runtime.h"
#include "../../include/parallel_func.h"

__global__ void parallel_add(int size, float *array1, float *array2, float *result)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < size; i += stride)
		result[i] = array1[i] + array2[i];
}

__global__ void parallel_mul(int size, float *array1, float *array2, float *result)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < size; i += stride)
		result[i] = array1[i] * array2[i];
}

__global__ void parallel_mul(int size, float *array, float scalar, float *result)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < size; i += stride)
		result[i] = array[i] * scalar;
}